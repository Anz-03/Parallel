#include "hip/hip_runtime.h"
//
// Created by Liu Yuxuan on 2024/9/11
// Modified from Zhong Yebin's PartB on 2023/9/16
//
// Email: yebinzhong@link.cuhk.edu.cn
// Email: yuxuanliu1@link.cuhk.edu.cn
//
// CUDA implementation of bilateral filtering on JPEG image
//

#include <iostream>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>

#include "../utils.hpp"
using namespace std;
/**
 * Demo kernel device function to clamp pixel value
 * 
 * You may mimic this to implement your own kernel device functions
 */
__device__ unsigned char d_clamp_pixel_value(float value)
{
    return value > 255 ? 255
           : value < 0 ? 0
                       : static_cast<unsigned char>(value);
}

__device__ float d_bilateral_filter(const ColorValue* values, int row, int col,
                            int width)
{
    ColorValue value_11 = values[(row - 1) * width + (col - 1)];
    ColorValue value_12 = values[(row - 1) * width + col];
    ColorValue value_13 = values[(row - 1) * width + (col + 1)];
    ColorValue value_21 = values[row * width + (col - 1)];
    ColorValue value_22 = values[row * width + col];
    ColorValue value_23 = values[row * width + (col + 1)];
    ColorValue value_31 = values[(row + 1) * width + (col - 1)];
    ColorValue value_32 = values[(row + 1) * width + col];
    ColorValue value_33 = values[(row + 1) * width + (col + 1)];
    // Spatial Weights
    float w_spatial_border = expf(-1 / 2 * powf_SIGMA_D_2);
    float w_spatial_corner = expf(2 * -1 / 2 * powf_SIGMA_D_2);
    // Intensity Weights
    ColorValue center_value = value_22;
    float w_11 = w_spatial_corner * expf(powf(center_value - value_11, 2) *
                                         minus_half_powf_SIGMA_R_2);
    float w_12 = w_spatial_border * expf(powf(center_value - value_12, 2) *
                                         minus_half_powf_SIGMA_R_2);
    float w_13 = w_spatial_corner * expf(powf(center_value - value_13, 2) *
                                         minus_half_powf_SIGMA_R_2);
    float w_21 = w_spatial_border * expf(powf(center_value - value_21, 2) *
                                         minus_half_powf_SIGMA_R_2);
    float w_22 = 1.0;
    float w_23 = w_spatial_border * expf(powf(center_value - value_23, 2) *
                                         minus_half_powf_SIGMA_R_2);
    float w_31 = w_spatial_corner * expf(powf(center_value - value_31, 2) *
                                         minus_half_powf_SIGMA_R_2);
    float w_32 = w_spatial_border * expf(powf(center_value - value_32, 2) *
                                         minus_half_powf_SIGMA_R_2);
    float w_33 = w_spatial_border * expf(powf(center_value - value_33, 2) *
                                         minus_half_powf_SIGMA_R_2);
    float sum_weights =
        w_11 + w_12 + w_13 + w_21 + w_22 + w_23 + w_31 + w_32 + w_33;
    // Calculate filtered value
    float filtered_value =
        (w_11 * value_11 + w_12 * value_12 + w_13 * value_13 + w_21 * value_21 +
         w_22 * center_value + w_23 * value_23 + w_31 * value_31 +
         w_32 * value_32 + w_33 * value_33) /
        sum_weights;
    return d_clamp_pixel_value(filtered_value);
}


__global__ void apply_filter_kernel(unsigned char* inputRvalues,
    unsigned char* inputGvalues,
    unsigned char* inputBvalues,
    unsigned char* outputRChannel,
    unsigned char* outputGChannel,
    unsigned char* outputBChannel,
    int width,
    int height,
    int start_row,
    int end_row)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 1 && x < width - 1 && y >= 1 && y < height - 1)
    {
            int id = y * width + x;

            // ColorValue r_sum = d_bilateral_filter(inputRvalues, y, x, width);
            // ColorValue g_sum = d_bilateral_filter(inputBvalues, y, x, width);
            // ColorValue b_sum = d_bilateral_filter(inputGvalues, y, x, width);
            ColorValue r_sum = d_bilateral_filter(inputRvalues, y, x, width);
            ColorValue g_sum = d_bilateral_filter(inputGvalues, y, x, width);
            ColorValue b_sum = d_bilateral_filter(inputBvalues, y, x, width); // 交换过绿色和蓝色就正常
            

            outputRChannel[id] = r_sum;
            outputGChannel[id] = g_sum;
            outputBChannel[id] = b_sum;
    }
}

int main(int argc, char** argv)
{
    if (argc != 3)
    {
        std::cerr << "Invalid argument, should be: ./executable "
                     "/path/to/input/jpeg /path/to/output/jpeg\n";
        return -1;
    }
    // Read input JPEG image in structure-of-array form
    const char* input_filename = argv[1];
    std::cout << "Input file from: " << input_filename << "\n";
    JpegSOA input_jpeg = read_jpeg_soa(input_filename);
    // if (input_jpeg.r_values == nullptr)
    // {
    //     std::cerr << "Failed to read input JPEG image\n";
    //     return -1;
    // }

    // Apply the filter to the image
    size_t buffer_size =
        input_jpeg.width * input_jpeg.height;
    unsigned char* filteredImageR = new unsigned char[buffer_size];
    unsigned char* filteredImageG = new unsigned char[buffer_size];
    unsigned char* filteredImageB = new unsigned char[buffer_size];

    // Allocate GPU memory
    unsigned char* d_input_buffer_r;
    unsigned char* d_input_buffer_g;
    unsigned char* d_input_buffer_b;
    unsigned char* d_filtered_imageR;
    unsigned char* d_filtered_imageG;
    unsigned char* d_filtered_imageB;
    float(*d_filter)[FILTERSIZE];

    hipMalloc((void**)&d_input_buffer_r, buffer_size);
    hipMalloc((void**)&d_input_buffer_g, buffer_size);
    hipMalloc((void**)&d_input_buffer_b, buffer_size);
    hipMalloc((void**)&d_filtered_imageR, buffer_size);
    hipMalloc((void**)&d_filtered_imageG, buffer_size);
    hipMalloc((void**)&d_filtered_imageB, buffer_size);
    hipMalloc((void**)&d_filter, FILTERSIZE * FILTERSIZE * sizeof(float));

    hipMemset(d_filtered_imageR, 0, buffer_size);
    hipMemset(d_filtered_imageG, 0, buffer_size);
    hipMemset(d_filtered_imageB, 0, buffer_size);

    // Copy input data from host to device
    hipMemcpy(d_input_buffer_r, input_jpeg.r_values,buffer_size,
               hipMemcpyHostToDevice);
    hipMemcpy(d_input_buffer_g, input_jpeg.g_values,buffer_size,
               hipMemcpyHostToDevice);
    hipMemcpy(d_input_buffer_b, input_jpeg.b_values,buffer_size,
               hipMemcpyHostToDevice);

    // Set CUDA grid and block sizes
    dim3 blockDim(32, 32);
    dim3 gridDim((input_jpeg.width + blockDim.x - 1) / blockDim.x,
                 (input_jpeg.height + blockDim.y - 1) / blockDim.y);

    hipEvent_t start, stop;
    float gpuDuration;
    hipEventCreate(&start); 
    hipEventCreate(&stop);
    // Perform filtering on GPU
    hipEventRecord(start, 0); // GPU start time
    // Launch CUDA kernel
    apply_filter_kernel<<<gridDim, blockDim>>>(
        d_input_buffer_r,d_input_buffer_g,d_input_buffer_b,
        d_filtered_imageR,d_filtered_imageG,d_filtered_imageB,
        // d_input_buffer_r,d_input_buffer_g,d_input_buffer_b,
        input_jpeg.width,input_jpeg.height,0,input_jpeg.height);
    hipEventRecord(stop, 0); // GPU end time
    hipEventSynchronize(stop);
    // Print the result of the GPU computation
    hipEventElapsedTime(&gpuDuration, start, stop);
    // Copy output data from GPU
    hipMemcpy(filteredImageR, d_filtered_imageR, buffer_size,
               hipMemcpyDeviceToHost);
    hipMemcpy(filteredImageG, d_filtered_imageG, buffer_size,
               hipMemcpyDeviceToHost);
    hipMemcpy(filteredImageB, d_filtered_imageB, buffer_size,
               hipMemcpyDeviceToHost);

    // Save output JPEG image
    const char* output_filepath = argv[2];
        std::cout << "Output file to: " << output_filepath << "\n";
        JpegSOA output_jpeg{filteredImageR,filteredImageG,filteredImageB,
        input_jpeg.width, input_jpeg.height,
        input_jpeg.num_channels, input_jpeg.color_space};
    if (export_jpeg(output_jpeg, output_filepath))
    {
        std::cerr << "Failed to write output JPEG\n";
        return -1;
    }
    // Post-processing
    // delete[] input_jpeg.buffer;
    // delete[] filteredImage;
    // Release GPU memory
    hipFree(d_input_buffer_r);
    hipFree(d_input_buffer_g);
    hipFree(d_input_buffer_b);
    hipFree(d_filtered_imageR);
    hipFree(d_filtered_imageG);
    hipFree(d_filtered_imageB);
    hipFree(d_filter);
    std::cout << "Transformation Complete!" << std::endl;
    std::cout << "GPU Execution Time: " << gpuDuration << " milliseconds"
              << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
